//
// Created by niceme on 2023/3/30.
//


#include <hip/hip_runtime.h>
#include <iostream>



__global__ void preprocess_kernel(float *data, const float *image_data, int input_size, int channelLength, const float *img_mean, const float *img_std) {
//    unsigned int c = blockIdx.x;
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.y;

    if (idx < channelLength) {
        int row = idx / input_size;
        int col = idx % input_size;
//        int offset = c * channelLength + row * input_size + col;
        int offset = idx;

        float pixel = image_data[offset];
        data[offset] = (pixel - img_mean[0]) * img_std[0];
    }
}


int main() {
    std::string engine_file_path = "/home/nvidia/nanodet-plus-m_416.engine";

    return 0;
}